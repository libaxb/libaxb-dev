#include <stdlib.h>
#include <string.h>
#include <stdio.h>
#include "libaxb.h"
#include "libaxb/backend.h"

#include <hip/hip_runtime.h>


static void * cuda_malloc(size_t size_in_bytes, void *aux_data)
{
  (void)aux_data;
  void *dev_ptr;
  hipError_t err = hipMalloc(&dev_ptr, size_in_bytes);
  if (err) return NULL;
  //printf("Allocated device ptr %p\n", dev_ptr);
  return dev_ptr;
}

static axbStatus_t cuda_free(void *ptr_to_free, void *aux_data)
{
  (void)aux_data;
  hipFree(ptr_to_free);
  return 0;
}


static axbStatus_t host_copyin(void *src, axbDataType_t src_type, void *dest, axbDataType_t dest_type, size_t n)
{
  if (src_type != AXB_REAL_DOUBLE || dest_type != AXB_REAL_DOUBLE) return 17590; // not yet supported

  //printf("Calling CUDA memcopy host to device %p\n", dest);
  return hipMemcpy(dest, src, sizeof(double) * n, hipMemcpyHostToDevice);
}

static axbStatus_t host_copyout(void *src, axbDataType_t src_type, void *dest, axbDataType_t dest_type, size_t n)
{
  if (src_type != AXB_REAL_DOUBLE || dest_type != AXB_REAL_DOUBLE) return 17590; // not yet supported
  //printf("Calling CUDA memcopy device %p to host\n", src);
  return hipMemcpy(dest, src, sizeof(double) * n, hipMemcpyDeviceToHost);
}

extern "C" axbStatus_t axbMemBackendRegister_CUDA(axbHandle_t handle)
{
  axbMemBackend_t cuda_backend;
  axbMemBackendCreate(&cuda_backend);

  // populate host_backend:
  axbMemBackendSetName(cuda_backend, "CUDA");
  axbMemBackendSetMalloc(cuda_backend, cuda_malloc);
  axbMemBackendSetFree(cuda_backend, cuda_free);

  axbMemBackendSetCopyIn(cuda_backend, host_copyin);
  axbMemBackendSetCopyOut(cuda_backend, host_copyout);

  // push into enclosing context identified by handle:
  axbMemBackendRegister(handle, cuda_backend);
  return 0;
}

