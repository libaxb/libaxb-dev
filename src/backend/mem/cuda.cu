#include <stdlib.h>
#include <string.h>
#include <stdio.h>
#include "libaxb.h"
#include "libaxb/backend.h"
#include <assert.h>

#include <hip/hip_runtime.h>


static axbStatus_t cuda_malloc(void **ptr, size_t size_in_bytes, void *aux_data)
{
  (void)aux_data;
  hipError_t err = hipMalloc(ptr, size_in_bytes);
  //printf("Allocated device ptr %p\n", *ptr);
  assert(err == 0 && "hipMalloc failed!");
  return err;
}

static axbStatus_t cuda_free(void *ptr_to_free, void *aux_data)
{
  (void)aux_data;
  hipFree(ptr_to_free);
  return 0;
}


static axbStatus_t cuda_copyin(void *src, axbDataType_t src_type, void *dest, axbDataType_t dest_type, size_t n, void *aux_data)
{
  if (src_type != AXB_REAL_DOUBLE || dest_type != AXB_REAL_DOUBLE) return 17590; // not yet supported

  (void)aux_data;
  return hipMemcpy(dest, src, sizeof(double) * n, hipMemcpyHostToDevice);
}

static axbStatus_t cuda_copyout(void *src, axbDataType_t src_type, void *dest, axbDataType_t dest_type, size_t n, void *aux_data)
{
  if (src_type != AXB_REAL_DOUBLE || dest_type != AXB_REAL_DOUBLE) return 17590; // not yet supported

  (void)aux_data;
  return hipMemcpy(dest, src, sizeof(double) * n, hipMemcpyDeviceToHost);
}

extern "C" axbStatus_t axbMemBackendRegister_CUDA(struct axbHandle_s *handle)
{
  struct axbMemBackend_s *cuda_backend;
  axbMemBackendCreate(&cuda_backend);

  // populate host_backend:
  axbMemBackendSetName(cuda_backend, "CUDA");
  axbMemBackendSetMalloc(cuda_backend, cuda_malloc);
  axbMemBackendSetFree(cuda_backend, cuda_free);

  axbMemBackendSetCopyIn(cuda_backend, cuda_copyin);
  axbMemBackendSetCopyOut(cuda_backend, cuda_copyout);

  // push into enclosing context identified by handle:
  axbMemBackendRegister(handle, cuda_backend);
  return 0;
}

