#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include "libaxb.h"
#include "libaxb/backend.h"
#include "libaxb/general.h"

#include <hip/hip_runtime.h>

__global__
void kernel_axpy(int n, double *y, const double *alpha, const double *x)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = *alpha * x[i] + y[i];
}


static axbStatus_t op_axpy(axbVec_t y, axbScalar_t alpha, axbVec_t x, void *aux_data)
{
  (void)aux_data;

  double *d_y     = (double*)y->data;
  double *d_alpha = (double*)alpha->data;
  double *d_x     = (double*)x->data;

  kernel_axpy<<<256, 256>>>((int)y->size, d_y, d_alpha, d_x);

  return 0;
}




extern "C" axbStatus_t axbOpBackendRegister_CUDA(axbHandle_t handle)
{
  axbOpBackend_t cuda_backend;
  axbOpBackendCreate(&cuda_backend);

  // populate host_backend:
  axbOpBackendSetName(cuda_backend, "CUDA");

  axbOperationID_t op_id = 0;
  axbOpBackendAddOperation(cuda_backend, "vec-axpy", (axbStatus_t (*)(void))op_axpy, NULL, &op_id);

  // push into enclosing context identified by handle:
  axbOpBackendRegister(handle, cuda_backend);

  return 0;
}

