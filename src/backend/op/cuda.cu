#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include "libaxb.h"
#include "libaxb/backend.h"
#include "libaxb/general.h"

#include <hip/hip_runtime.h>

__global__
void kernel_axpy(int n, double *y, const double *alpha, const double *x)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = *alpha * x[i] + y[i];
}


static axbStatus_t op_axpy(axbVec_t y, axbScalar_t alpha, axbVec_t x, void *aux_data)
{
  (void)aux_data;

  double *d_y     = (double*)y->data;
  double *d_alpha = (double*)alpha->data;
  double *d_x     = (double*)x->data;

  kernel_axpy<<<256, 256>>>((int)y->size, d_y, d_alpha, d_x);

  return 0;
}




extern "C" axbStatus_t axbOpBackendRegister_CUDA(axbHandle_t handle)
{
  axbOpBackend_t cuda_backend;
  axbStatus_t status = axbOpBackendCreate(&cuda_backend); AXB_ERRCHK(status);

  // populate host_backend:
  status = axbOpBackendSetName(cuda_backend, "CUDA"); AXB_ERRCHK(status);

  axbOperationID_t op_id = 0;
  status = axbOpBackendAddOperation(cuda_backend, "vec-axpy", (axbStatus_t (*)(void))op_axpy, NULL, &op_id); AXB_ERRCHK(status);

  // push into enclosing context identified by handle:
  status = axbOpBackendRegister(handle, cuda_backend); AXB_ERRCHK(status);

  return 0;
}

