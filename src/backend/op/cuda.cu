#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include "libaxb.h"
#include "libaxb/backend.h"
#include "libaxb/general.h"
#include "libaxb/backend/op.h"

#include <hip/hip_runtime.h>

/////////////////////

__global__
static void kernel_vec_set_from_device(int n, double *x, const double *alpha)
{
  for (int i = blockIdx.x*blockDim.x + threadIdx.x; i<n; i += gridDim.x * blockDim.x) x[i] = *alpha;
}

__global__
static void kernel_vec_set_from_host(int n, double *x, double alpha)
{
  for (int i = blockIdx.x*blockDim.x + threadIdx.x; i<n; i += gridDim.x * blockDim.x) x[i] = alpha;
}


static axbStatus_t op_vec_set(axbVec_t x, axbScalar_t alpha, void *aux_data)
{
  (void)aux_data;

  double *d_x     = (double*)x->data;

  if (strcmp(alpha->memBackend->name, "host") != 0) {  // alpha on GPU
    double *d_alpha = (double*)alpha->data;
    kernel_vec_set_from_device<<<256, 256>>>((int)x->size, d_x, d_alpha);
  } else { // alpha on CPU
    double d_alpha = *((double*)alpha->data);
    kernel_vec_set_from_host<<<256, 256>>>((int)x->size, d_x, d_alpha);
  }

  return 0;
}

/////////////////////

__global__
static void kernel_vec_sqrtabs(int n, double *x)
{
  for (int i = blockIdx.x*blockDim.x + threadIdx.x; i<n; i += gridDim.x * blockDim.x) x[i] = sqrt(fabs(x[i]));
}


static axbStatus_t op_vec_sqrtabs(axbVec_t x, void *aux_data)
{
  (void)aux_data;

  double *d_x     = (double*)x->data;
  kernel_vec_sqrtabs<<<256, 256>>>((int)x->size, d_x);

  return 0;
}

/////////////////////

static axbStatus_t op_vec_zero(axbVec_t x, void *aux_data)
{
  (void)aux_data;

  double *d_x     = (double*)x->data;
  kernel_vec_set_from_host<<<256, 256>>>((int)x->size, d_x, 0);

  return 0;
}

/////////////////////

__global__
static void kernel_vec_scale_from_device(int n, double *x, const double *alpha)
{
  for (int i = blockIdx.x*blockDim.x + threadIdx.x; i<n; i += gridDim.x * blockDim.x) x[i] *= *alpha;
}

__global__
static void kernel_vec_scale_from_host(int n, double *x, double alpha)
{
  for (int i = blockIdx.x*blockDim.x + threadIdx.x; i<n; i += gridDim.x * blockDim.x) x[i] *= alpha;
}


static axbStatus_t op_vec_scale(axbVec_t x, axbScalar_t alpha, void *aux_data)
{
  (void)aux_data;

  double *d_x     = (double*)x->data;

  if (strcmp(alpha->memBackend->name, "host") != 0) {  // alpha on GPU
    double *d_alpha = (double*)alpha->data;
    kernel_vec_scale_from_device<<<256, 256>>>((int)x->size, d_x, d_alpha);
  } else { // alpha on CPU
    double d_alpha = *((double*)alpha->data);
    kernel_vec_scale_from_host<<<256, 256>>>((int)x->size, d_x, d_alpha);
  }

  return 0;
}

//
// Reduction operations
//

#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 600
__device__ double atomicAdd(double* address, double val) {
  unsigned long long int* address_as_ull = (unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;
  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
  } while (assumed != old);
  return __longlong_as_double(old);
}
#endif

/////////////////////

__global__
static void kernel_vec_sum(int n, double *x, double *alpha)
{
  __shared__ double reduction_buffer[256];
  double t = 0;
  for (int i = blockIdx.x*blockDim.x + threadIdx.x; i<n; i += gridDim.x * blockDim.x) t += x[i];

  reduction_buffer[threadIdx.x] = t;

  // parallel reduction
  for (unsigned int stride = blockDim.x/2; stride > 0; stride /= 2)
  {
    __syncthreads();
    if (threadIdx.x < stride)
      reduction_buffer[threadIdx.x] += reduction_buffer[threadIdx.x+stride];
  }

  if (threadIdx.x == 0)
    atomicAdd(alpha, reduction_buffer[0]);
}


static axbStatus_t op_vec_sum(axbVec_t x, axbScalar_t alpha, void *aux_data)
{
  (void)aux_data;

  double *d_x     = (double*)x->data;
  double *d_alpha = (double*)alpha->data;
  kernel_vec_set_from_host<<<1, 1>>>((int)1, d_alpha, 0);
  kernel_vec_sum<<<256, 256>>>((int)x->size, d_x, d_alpha);

  return 0;
}

/////////////////////

__global__
static void kernel_vec_dot(int n, double *x, double *y, double *alpha)
{
  __shared__ double reduction_buffer[256];
  double t = 0;
  for (int i = blockIdx.x*blockDim.x + threadIdx.x; i<n; i += gridDim.x * blockDim.x) t += x[i] * y[i];

  reduction_buffer[threadIdx.x] = t;

  // parallel reduction
  for (unsigned int stride = blockDim.x/2; stride > 0; stride /= 2)
  {
    __syncthreads();
    if (threadIdx.x < stride)
      reduction_buffer[threadIdx.x] += reduction_buffer[threadIdx.x+stride];
  }

  if (threadIdx.x == 0)
    atomicAdd(alpha, reduction_buffer[0]);
}


static axbStatus_t op_vec_dot(axbVec_t x, axbVec_t y, axbScalar_t alpha, void *aux_data)
{
  (void)aux_data;

  double *d_x     = (double*)x->data;
  double *d_y     = (double*)y->data;
  double *d_alpha = (double*)alpha->data;
  kernel_vec_set_from_host<<<1, 1>>>((int)1, d_alpha, 0);
  kernel_vec_dot<<<256, 256>>>((int)x->size, d_x, d_y, d_alpha);

  return 0;
}

/////////////////////

static axbStatus_t op_vec_tdot(axbVec_t x, axbVec_t y, axbScalar_t alpha, void *aux_data)
{
  return op_vec_dot(x, y, alpha, aux_data); // TODO: update for complex scalar types
}

/////////////////////

static axbStatus_t op_vec_mdot(axbVec_t x, size_t num_vecs, const axbVec_t *y, axbScalar_t *mdot, void *aux_data)
{
  (void)aux_data;

  // TODO: Replace by faster variant
  for (size_t i=0; i<num_vecs; ++i)
    op_vec_dot(x, y[i], mdot[i], aux_data);

  return 0;
}

/////////////////////

__global__
static void kernel_vec_norm1(int n, double *x, double *alpha)
{
  __shared__ double reduction_buffer[256];
  double t = 0;
  for (int i = blockIdx.x*blockDim.x + threadIdx.x; i<n; i += gridDim.x * blockDim.x) t += fabs(x[i]);

  reduction_buffer[threadIdx.x] = t;

  // parallel reduction
  for (unsigned int stride = blockDim.x/2; stride > 0; stride /= 2)
  {
    __syncthreads();
    if (threadIdx.x < stride)
      reduction_buffer[threadIdx.x] += reduction_buffer[threadIdx.x+stride];
  }

  if (threadIdx.x == 0)
    atomicAdd(alpha, reduction_buffer[0]);
}


static axbStatus_t op_vec_norm1(axbVec_t x, axbScalar_t alpha, void *aux_data)
{
  (void)aux_data;

  double *d_x     = (double*)x->data;
  double *d_alpha = (double*)alpha->data;
  kernel_vec_set_from_host<<<1, 1>>>((int)1, d_alpha, 0);
  kernel_vec_norm1<<<256, 256>>>((int)x->size, d_x, d_alpha);

  return 0;
}

/////////////////////

__global__
static void kernel_vec_norm2(int n, double *x, double *alpha)
{
  __shared__ double reduction_buffer[256];
  double t = 0;
  for (int i = blockIdx.x*blockDim.x + threadIdx.x; i<n; i += gridDim.x * blockDim.x) t += x[i] * x[i];

  reduction_buffer[threadIdx.x] = t;

  // parallel reduction
  for (unsigned int stride = blockDim.x/2; stride > 0; stride /= 2)
  {
    __syncthreads();
    if (threadIdx.x < stride)
      reduction_buffer[threadIdx.x] += reduction_buffer[threadIdx.x+stride];
  }

  if (threadIdx.x == 0)
    atomicAdd(alpha, reduction_buffer[0]);
}


static axbStatus_t op_vec_norm2(axbVec_t x, axbScalar_t alpha, void *aux_data)
{
  (void)aux_data;

  double *d_x     = (double*)x->data;
  double *d_alpha = (double*)alpha->data;
  kernel_vec_set_from_host<<<1, 1>>>((int)1, d_alpha, 0);
  kernel_vec_norm2<<<256, 256>>>((int)x->size, d_x, d_alpha);
  kernel_vec_sqrtabs<<<1, 1>>>((int)1, d_alpha);
  return 0;
}

/////////////////////

__global__
static void kernel_vec_norminf(int n, double *x, double *alpha)
{
  __shared__ double reduction_buffer[256];
  double t = 0;
  for (int i = blockIdx.x*blockDim.x + threadIdx.x; i<n; i += gridDim.x * blockDim.x) t = max(t, fabs(x[i]));

  reduction_buffer[threadIdx.x] = t;

  // parallel reduction
  for (unsigned int stride = blockDim.x/2; stride > 0; stride /= 2)
  {
    __syncthreads();
    if (threadIdx.x < stride)
      reduction_buffer[threadIdx.x] = max(reduction_buffer[threadIdx.x], reduction_buffer[threadIdx.x+stride]);
  }

  if (threadIdx.x == 0)
    alpha[blockIdx.x] = reduction_buffer[0];
}


static axbStatus_t op_vec_norminf(axbVec_t x, axbScalar_t alpha, void *aux_data)
{
  (void)aux_data;

  double *tmp;
  hipMalloc((void**)&tmp, sizeof(double) * 256);   // TODO: Avoid allocation in each call to op_vec_norminf

  double *d_x     = (double*)x->data;
  double *d_alpha = (double*)alpha->data;
  kernel_vec_set_from_host<<<1, 1>>>((int)1, d_alpha, 0);
  kernel_vec_norminf<<<256, 256>>>((int)x->size, d_x, tmp);
  kernel_vec_norminf<<<1, 256>>>((int)256, tmp, d_alpha);

  hipFree(tmp);

  return 0;
}

/////////////////////

__global__
static void kernel_vec_dotnorm2(int n, double *s, double *t, double *dot_st, double *norm_t)
{
  __shared__ double reduction_buffer[256];
  double dot = 0;
  double norm = 0;
  *dot_st = 0;
  *norm_t = 0;

  for (int i = blockIdx.x*blockDim.x + threadIdx.x; i<n; i += gridDim.x * blockDim.x) {
    double val_t = t[i];
    dot += s[i] * val_t;
    norm += val_t * val_t;
  }

  //
  // first reduction for dot
  //
  reduction_buffer[threadIdx.x] = dot;

  // parallel reduction
  for (unsigned int stride = blockDim.x/2; stride > 0; stride /= 2)
  {
    __syncthreads();
    if (threadIdx.x < stride)
      reduction_buffer[threadIdx.x] += reduction_buffer[threadIdx.x+stride];
  }

  if (threadIdx.x == 0)
    atomicAdd(dot_st, reduction_buffer[0]);

  //
  // second reduction for norm
  //

  reduction_buffer[threadIdx.x] = norm;

  // parallel reduction
  for (unsigned int stride = blockDim.x/2; stride > 0; stride /= 2)
  {
    __syncthreads();
    if (threadIdx.x < stride)
      reduction_buffer[threadIdx.x] += reduction_buffer[threadIdx.x+stride];
  }

  if (threadIdx.x == 0)
    atomicAdd(norm_t, reduction_buffer[0]);
}


static axbStatus_t op_vec_dotnorm2(axbVec_t s, axbVec_t t, axbScalar_t dot, axbScalar_t norm, void *aux_data)
{
  (void)aux_data;

  double *d_s     = (double*)s->data;
  double *d_t     = (double*)t->data;
  double *d_dot   = (double*)dot->data;
  double *d_norm  = (double*)norm->data;
  kernel_vec_set_from_host<<<1, 1>>>((int)1, d_dot, 0);
  kernel_vec_set_from_host<<<1, 1>>>((int)1, d_norm, 0);
  kernel_vec_dotnorm2<<<256, 256>>>((int)s->size, d_s, d_t, d_dot, d_norm);
  kernel_vec_sqrtabs<<<1, 1>>>((int)1, d_norm);

  return 0;
}

/////////////////////

__global__
static void kernel_vec_max(int n, double *x, int *index, double *alpha)
{
  __shared__ double reduction_buffer_max[256];
  __shared__ int    reduction_buffer_idx[256];
  double t = x[0];
  int idx = 0;
  for (int i = blockIdx.x*blockDim.x + threadIdx.x; i<n; i += gridDim.x * blockDim.x) {
    double xi = x[i];
    if (t < xi) {
      t = xi;
      idx = i;
    }
  }

  reduction_buffer_max[threadIdx.x] = t;
  reduction_buffer_idx[threadIdx.x] = idx;

  // parallel reduction
  for (unsigned int stride = blockDim.x/2; stride > 0; stride /= 2)
  {
    __syncthreads();
    if (threadIdx.x < stride) {
      if (reduction_buffer_max[threadIdx.x] < reduction_buffer_max[threadIdx.x+stride]) {
        reduction_buffer_max[threadIdx.x] = reduction_buffer_max[threadIdx.x+stride];
        reduction_buffer_idx[threadIdx.x] = reduction_buffer_idx[threadIdx.x+stride];
      }
    }
  }

  if (threadIdx.x == 0) {
    alpha[blockIdx.x] = reduction_buffer_max[0];
    index[blockIdx.x] = reduction_buffer_idx[0];
  }
}


static axbStatus_t op_vec_max(axbVec_t x, size_t *idx, axbScalar_t alpha, void *aux_data)
{
  (void)aux_data;

  //
  // TODO: Refactor this! The whole computation can be done without host<->device copies!
  //

  double *tmp;   hipMalloc((void**)&tmp,   sizeof(double) * 256);   // TODO: Avoid allocation in each call to op_vec_max
  int    *index; hipMalloc((void**)&index, sizeof(int)    * 256);   // TODO: Avoid allocation in each call to op_vec_max

  double *d_x     = (double*)x->data;
  double *d_alpha = (double*)alpha->data;
  kernel_vec_max<<<256, 256>>>((int)x->size, d_x, index, tmp);

  double host_val[256];
  hipMemcpy(host_val, tmp,   256 * sizeof(double), hipMemcpyDeviceToHost);

  int host_idx[256];
  hipMemcpy(host_idx, index, 256 * sizeof(int), hipMemcpyDeviceToHost);

  double val_max = host_val[0];
  int    idx_max = host_idx[0];
  for (size_t i=1; i<256; ++i) {
    if (val_max < host_val[i]) {
      val_max = host_val[i];
      idx_max = host_idx[i];
    }
  }
  *idx = idx_max;

  hipMemcpy(d_alpha, (void*)&val_max, sizeof(double), hipMemcpyHostToDevice);

  hipFree(tmp);
  hipFree(index);

  return 0;
}

/////////////////////

__global__
static void kernel_vec_min(int n, double *x, int *index, double *alpha)
{
  __shared__ double reduction_buffer_min[256];
  __shared__ int    reduction_buffer_idx[256];
  double t = x[0];
  int idx = 0;
  for (int i = blockIdx.x*blockDim.x + threadIdx.x; i<n; i += gridDim.x * blockDim.x) {
    double xi = x[i];
    if (t > xi) {
      t = xi;
      idx = i;
    }
  }

  reduction_buffer_min[threadIdx.x] = t;
  reduction_buffer_idx[threadIdx.x] = idx;

  // parallel reduction
  for (unsigned int stride = blockDim.x/2; stride > 0; stride /= 2)
  {
    __syncthreads();
    if (threadIdx.x < stride) {
      if (reduction_buffer_min[threadIdx.x] > reduction_buffer_min[threadIdx.x+stride]) {
        reduction_buffer_min[threadIdx.x] = reduction_buffer_min[threadIdx.x+stride];
        reduction_buffer_idx[threadIdx.x] = reduction_buffer_idx[threadIdx.x+stride];
      }
    }
  }

  if (threadIdx.x == 0) {
    alpha[blockIdx.x] = reduction_buffer_min[0];
    index[blockIdx.x] = reduction_buffer_idx[0];
  }
}


static axbStatus_t op_vec_min(axbVec_t x, size_t *idx, axbScalar_t alpha, void *aux_data)
{
  (void)aux_data;

  //
  // TODO: Refactor this! The whole computation can be done without host<->device copies!
  //

  double *tmp;   hipMalloc((void**)&tmp,   sizeof(double) * 256);   // TODO: Avoid allocation in each call to op_vec_max
  int    *index; hipMalloc((void**)&index, sizeof(int)    * 256);   // TODO: Avoid allocation in each call to op_vec_max

  double *d_x     = (double*)x->data;
  double *d_alpha = (double*)alpha->data;
  kernel_vec_min<<<256, 256>>>((int)x->size, d_x, index, tmp);

  double host_val[256];
  hipMemcpy(host_val, tmp,   256 * sizeof(double), hipMemcpyDeviceToHost);

  int host_idx[256];
  hipMemcpy(host_idx, index, 256 * sizeof(int), hipMemcpyDeviceToHost);

  double val_min = host_val[0];
  int    idx_min = host_idx[0];
  for (size_t i=1; i<256; ++i) {
    if (val_min > host_val[i]) {
      val_min = host_val[i];
      idx_min = host_idx[i];
    }
  }
  *idx = idx_min;

  hipMemcpy(d_alpha, (void*)&val_min, sizeof(double), hipMemcpyHostToDevice);

  hipFree(tmp);
  hipFree(index);

  return 0;
}


//
// Vector-vector operations
//

__global__
static void kernel_vec_copy(int n, double *x, double *y)
{
  for (int i = blockIdx.x*blockDim.x + threadIdx.x; i<n; i += gridDim.x * blockDim.x) y[i] = x[i];
}


static axbStatus_t op_vec_copy(axbVec_t x, axbVec_t y, void *aux_data)
{
  (void)aux_data;

  double *d_x     = (double*)x->data;
  double *d_y     = (double*)y->data;

  kernel_vec_copy<<<256, 256>>>((int)x->size, d_x, d_y);

  return 0;
}

/////////////////////

__global__
static void kernel_vec_swap(int n, double *x, double *y)
{
  for (int i = blockIdx.x*blockDim.x + threadIdx.x; i<n; i += gridDim.x * blockDim.x) {
    double t = y[i];
    y[i] = x[i];
    x[i] = t;
  }
}


static axbStatus_t op_vec_swap(axbVec_t x, axbVec_t y, void *aux_data)
{
  (void)aux_data;

  double *d_x     = (double*)x->data;
  double *d_y     = (double*)y->data;

  kernel_vec_swap<<<256, 256>>>((int)x->size, d_x, d_y);

  return 0;
}

/////////////////////

__global__
static void kernel_vec_axpy(int n, double *y, const double *alpha, const double *x)
{
  for (int i = blockIdx.x*blockDim.x + threadIdx.x; i<n; i += gridDim.x * blockDim.x) y[i] = *alpha * x[i] + y[i];
}


static axbStatus_t op_vec_axpy(axbVec_t y, axbScalar_t alpha, axbVec_t x, void *aux_data)
{
  (void)aux_data;

  double *d_y     = (double*)y->data;
  double *d_alpha = (double*)alpha->data;
  double *d_x     = (double*)x->data;

  kernel_vec_axpy<<<256, 256>>>((int)y->size, d_y, d_alpha, d_x);

  return 0;
}

/////////////////////

__global__
static void kernel_vec_aypx(int n, double *y, const double *alpha, const double *x)
{
  for (int i = blockIdx.x*blockDim.x + threadIdx.x; i<n; i += gridDim.x * blockDim.x) y[i] = *alpha * y[i] + x[i];
}


static axbStatus_t op_vec_aypx(axbVec_t y, axbScalar_t alpha, axbVec_t x, void *aux_data)
{
  (void)aux_data;

  double *d_y     = (double*)y->data;
  double *d_alpha = (double*)alpha->data;
  double *d_x     = (double*)x->data;

  kernel_vec_aypx<<<256, 256>>>((int)y->size, d_y, d_alpha, d_x);

  return 0;
}

/////////////////////

__global__
static void kernel_vec_axpbypcz(int n, double *z, const double *alpha, const double *beta, const double *gamma, const double *x, const double *y)
{
  for (int i = blockIdx.x*blockDim.x + threadIdx.x; i<n; i += gridDim.x * blockDim.x) z[i] = *alpha * x[i] + *beta * y[i] + *gamma * z[i];
}


static axbStatus_t op_vec_axpbypcz(axbVec_t z, axbScalar_t alpha, axbScalar_t beta, axbScalar_t gamma, axbVec_t x, axbVec_t y, void *aux_data)
{
  (void)aux_data;

  double *d_z     = (double*)z->data;
  double *d_alpha = (double*)alpha->data;
  double *d_beta  = (double*)beta->data;
  double *d_gamma = (double*)gamma->data;
  double *d_x     = (double*)x->data;
  double *d_y     = (double*)y->data;

  kernel_vec_axpbypcz<<<256, 256>>>((int)z->size, d_z, d_alpha, d_beta, d_gamma, d_x, d_y);

  return 0;
}

/////////////////////

__global__
static void kernel_vec_waxpy(int n, double *w, const double *alpha, const double *x, const double *y)
{
  for (int i = blockIdx.x*blockDim.x + threadIdx.x; i<n; i += gridDim.x * blockDim.x) w[i] = *alpha * x[i] + y[i];
}


static axbStatus_t op_vec_waxpy(axbVec_t w, axbScalar_t alpha, axbVec_t x, axbVec_t y, void *aux_data)
{
  (void)aux_data;

  double *d_w     = (double*)w->data;
  double *d_alpha = (double*)alpha->data;
  double *d_x     = (double*)x->data;
  double *d_y     = (double*)y->data;

  kernel_vec_waxpy<<<256, 256>>>((int)w->size, d_w, d_alpha, d_x, d_y);

  return 0;
}

/////////////////////

static axbStatus_t op_vec_maxpy(axbVec_t y, size_t num_vecs, const axbScalar_t *alpha, const axbVec_t *x, void *aux_data) {

  // TODO: Be more efficient than this!
  for (size_t i=0; i<num_vecs; ++i)
    op_vec_axpy(y, alpha[i], x[i], aux_data);
  return 0;
}

/////////////////////

__global__
static void kernel_vec_pointwisemult(int n, double *w, const double *x, const double *y)
{
  for (int i = blockIdx.x*blockDim.x + threadIdx.x; i<n; i += gridDim.x * blockDim.x) w[i] = x[i] * y[i];
}


static axbStatus_t op_vec_pointwisemult(axbVec_t w, axbVec_t x, axbVec_t y, void *aux_data)
{
  (void)aux_data;

  double *d_w     = (double*)w->data;
  double *d_x     = (double*)x->data;
  double *d_y     = (double*)y->data;

  kernel_vec_pointwisemult<<<256, 256>>>((int)w->size, d_w, d_x, d_y);

  return 0;
}

/////////////////////

__global__
static void kernel_vec_pointwisediv(int n, double *w, const double *x, const double *y)
{
  for (int i = blockIdx.x*blockDim.x + threadIdx.x; i<n; i += gridDim.x * blockDim.x) w[i] = x[i] / y[i];
}


static axbStatus_t op_vec_pointwisediv(axbVec_t w, axbVec_t x, axbVec_t y, void *aux_data)
{
  (void)aux_data;

  double *d_w     = (double*)w->data;
  double *d_x     = (double*)x->data;
  double *d_y     = (double*)y->data;

  kernel_vec_pointwisediv<<<256, 256>>>((int)w->size, d_w, d_x, d_y);

  return 0;
}

/////////////////////



extern "C" axbStatus_t axbOpBackendRegister_CUDA(axbHandle_t handle)
{
  axbOpBackend_t cuda_backend;
  axbStatus_t status = axbOpBackendCreate(&cuda_backend); AXB_ERRCHK(status);

  // populate host_backend:
  status = axbOpBackendSetName(cuda_backend, "CUDA"); AXB_ERRCHK(status);

  axbOperationID_t op_id = 0;

#define AXB_ADD_OPERATION(OPNAME, ENUMCONSTANT)    status = axbOpBackendAddOperation(cuda_backend, #OPNAME,     (axbStatus_t (*)(void))OPNAME,     NULL, &op_id); AXB_ERRCHK(status); assert(op_id == ENUMCONSTANT && "Logic error: op_id != " #ENUMCONSTANT)

  // inplace operations
  AXB_ADD_OPERATION(op_vec_set,     AXB_OP_VEC_SET);
  AXB_ADD_OPERATION(op_vec_sqrtabs, AXB_OP_VEC_SQRTABS);
  AXB_ADD_OPERATION(op_vec_zero,    AXB_OP_VEC_ZERO);
  AXB_ADD_OPERATION(op_vec_scale,   AXB_OP_VEC_SCALE);

  // reduction operations
  AXB_ADD_OPERATION(op_vec_sum,      AXB_OP_VEC_SUM);
  AXB_ADD_OPERATION(op_vec_dot,      AXB_OP_VEC_DOT);
  AXB_ADD_OPERATION(op_vec_tdot,     AXB_OP_VEC_TDOT);
  AXB_ADD_OPERATION(op_vec_mdot,     AXB_OP_VEC_MDOT);
  AXB_ADD_OPERATION(op_vec_norm1,    AXB_OP_VEC_NORM1);
  AXB_ADD_OPERATION(op_vec_norm2,    AXB_OP_VEC_NORM2);
  AXB_ADD_OPERATION(op_vec_norminf,  AXB_OP_VEC_NORMINF);
  AXB_ADD_OPERATION(op_vec_dotnorm2, AXB_OP_VEC_DOTNORM2);
  AXB_ADD_OPERATION(op_vec_max,      AXB_OP_VEC_MAX);
  AXB_ADD_OPERATION(op_vec_min,      AXB_OP_VEC_MIN);

  // vector-vector operations
  AXB_ADD_OPERATION(op_vec_copy,          AXB_OP_VEC_COPY);
  AXB_ADD_OPERATION(op_vec_swap,          AXB_OP_VEC_SWAP);
  AXB_ADD_OPERATION(op_vec_axpy,          AXB_OP_VEC_AXPY);
  AXB_ADD_OPERATION(op_vec_aypx,          AXB_OP_VEC_AYPX);
  AXB_ADD_OPERATION(op_vec_axpbypcz,      AXB_OP_VEC_AXPBYPCZ);
  AXB_ADD_OPERATION(op_vec_waxpy,         AXB_OP_VEC_WAXPY);
  AXB_ADD_OPERATION(op_vec_maxpy,         AXB_OP_VEC_MAXPY);
  AXB_ADD_OPERATION(op_vec_pointwisemult, AXB_OP_VEC_POINTWISEMULT);
  AXB_ADD_OPERATION(op_vec_pointwisediv,  AXB_OP_VEC_POINTWISEDIV);

#undef AXB_ADD_OPERATION

  // push into enclosing context identified by handle:
  status = axbOpBackendRegister(handle, cuda_backend); AXB_ERRCHK(status);

  return 0;
}

